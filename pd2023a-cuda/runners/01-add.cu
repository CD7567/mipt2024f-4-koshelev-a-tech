#include "KernelAdd.cuh"
#include "CommonKernels.cuh"
#include <cstdio>

int main(int argc, char** argv) {
    int SIZE = std::atoi(argv[1]);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float *x, *y, *result;
    float *dev_x, *dev_y, *dev_result;

    x = (float*) malloc(SIZE * sizeof(float));
    y = (float*) malloc(SIZE * sizeof(float));
    result = (float*) malloc(SIZE * sizeof(float));

    HANDLE_ERROR(hipMalloc((void**) &dev_x, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_y, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_result, SIZE * sizeof(float)));

    for (int i = 0; i < SIZE; ++i) {
        x[i] = i;
        y[i] = i;
    }

    HANDLE_ERROR(hipMemcpy(dev_x, x, SIZE * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_y, y, SIZE * sizeof(float), hipMemcpyHostToDevice));

    dim3 grid_size((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 block_size(BLOCK_SIZE);

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	hipEventRecord(begin);
    KernelAdd<<<grid_size, block_size>>>(SIZE, dev_x, dev_y, dev_result);
    hipEventRecord(end);

    hipEventSynchronize(end);

    FILE* csv_output = fopen("data/data.csv", "a");
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, begin, end);
    fprintf(csv_output, "KernelAdd, %d, %d, %f", SIZE, BLOCK_SIZE, elapsed_time);
    fclose(csv_output);

    HANDLE_ERROR(hipMemcpy(result, dev_result, SIZE * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_result);

    free(x);
    free(y);
    free(result);
}
