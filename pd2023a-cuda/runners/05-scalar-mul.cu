#include <ScalarMulRunner.cuh>
#include "CommonKernels.cuh"
#include <cstdio>

int main(int argc, char** argv) {
    int SIZE = std::atoi(argv[1]);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float *x, *y, *result;
    float *dev_x, *dev_y, *dev_result;

    x = (float*) malloc(SIZE * sizeof(float));
    y = (float*) malloc(SIZE * sizeof(float));
    result = (float*) malloc(SIZE * sizeof(float));

    HANDLE_ERROR(hipMalloc((void**) &dev_x, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_y, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_result, SIZE * sizeof(float)));

    for (int i = 0; i < SIZE; ++i) {
        x[i] = i;
        y[i] = i;
    }

    HANDLE_ERROR(hipMemcpy(dev_x, x, SIZE * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_y, y, SIZE * sizeof(float), hipMemcpyHostToDevice));

	  hipEvent_t begin_tr, end_tr, begin_sp, end_sp;
	  hipEventCreate(&begin_tr);
	  hipEventCreate(&end_tr);
	  hipEventCreate(&begin_sp);
	  hipEventCreate(&end_sp);

	  hipEventRecord(begin_tr);
    ScalarMulTwoReductions(SIZE, dev_x, dev_y, BLOCK_SIZE);
    hipEventRecord(end_tr);

    hipEventSynchronize(end_tr);

    hipEventRecord(begin_sp);
    ScalarMulSumPlusReduction(SIZE, dev_x, dev_y, BLOCK_SIZE);
    hipEventRecord(end_sp);

    hipEventSynchronize(end_sp);

    FILE* csv_output = fopen("data/data.csv", "a");
    float elapsed_time_tr, elapsed_time_sp;
    hipEventElapsedTime(&elapsed_time_tr, begin_tr, end_tr);
    hipEventElapsedTime(&elapsed_time_sp, begin_sp, end_sp);
    fprintf(csv_output, "ScalarMulTR, %d, %d, %f", SIZE, BLOCK_SIZE, elapsed_time_tr);
    fprintf(csv_output, "ScalarMulSP, %d, %d, %f", SIZE, BLOCK_SIZE, elapsed_time_sp);
    fclose(csv_output);

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_result);

    free(x);
    free(y);
    free(result);
}
