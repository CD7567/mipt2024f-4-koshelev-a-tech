#include <CosineVector.cuh>
#include "CommonKernels.cuh"
#include <cstdio>

int main(int argc, char** argv) {
    int SIZE = std::atoi(argv[1]);
    int BLOCK_SIZE = std::atoi(argv[2]);

    float *x, *y, *result;
    float *dev_x, *dev_y, *dev_result;

    x = (float*) malloc(SIZE * sizeof(float));
    y = (float*) malloc(SIZE * sizeof(float));
    result = (float*) malloc(SIZE * sizeof(float));

    HANDLE_ERROR(hipMalloc((void**) &dev_x, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_y, SIZE * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**) &dev_result, SIZE * sizeof(float)));

    for (int i = 0; i < SIZE; ++i) {
        x[i] = i;
        y[i] = i;
    }

    HANDLE_ERROR(hipMemcpy(dev_x, x, SIZE * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_y, y, SIZE * sizeof(float), hipMemcpyHostToDevice));

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	hipEventRecord(begin);
    CosineVector(SIZE, dev_x, dev_y, BLOCK_SIZE);
    hipEventRecord(end);

    hipEventSynchronize(end);

    FILE* csv_output = fopen("data/data.csv", "a");
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, begin, end);
    fprintf(csv_output, "CosineVector, %d, %d, %f", SIZE, BLOCK_SIZE, elapsed_time);
    fclose(csv_output);

    HANDLE_ERROR(hipMemcpy(result, dev_result, SIZE * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_result);

    free(x);
    free(y);
    free(result);
}

