#include "KernelMatrixAdd.cuh"
#include "CommonKernels.cuh"
#include <cstdio>
#include <random>

int main(int argc, char** argv) {
    int WIDTH = std::atoi(argv[1]);
    int HEIGHT = std::atoi(argv[2]);
    int LENGTH = WIDTH * HEIGHT;
    int BLOCK_SIZE = std::atoi(argv[3]);

    float *A, *B, *result;
    float *dev_A, *dev_B, *dev_result;

    size_t pitch;
    HANDLE_ERROR(hipMallocPitch(&dev_A, &pitch, WIDTH, HEIGHT));
    HANDLE_ERROR(hipMallocPitch(&dev_B, &pitch, WIDTH, HEIGHT));
    HANDLE_ERROR(hipMallocPitch(&dev_result, &pitch, WIDTH, HEIGHT));

    HANDLE_ERROR(hipMemcpy2D(dev_A, pitch, A, WIDTH * sizeof(float), WIDTH, HEIGHT, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy2D(dev_B, pitch, B, WIDTH * sizeof(float), WIDTH, HEIGHT, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy2D(dev_result, pitch, result, WIDTH * sizeof(float), WIDTH, HEIGHT, hipMemcpyHostToDevice));

    dim3 grid_size((WIDTH +  BLOCK_SIZE - 1) / WIDTH, (HEIGHT +  BLOCK_SIZE - 1) / HEIGHT);
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

    hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

    hipEventRecord(begin);
    KernelMatrixAdd<<<grid_size, block_size>>>(HEIGHT, WIDTH, pitch, dev_A, dev_B, dev_result);
    hipEventRecord(end);

	hipEventSynchronize(end);

	FILE* csv_output = fopen("data/data.csv", "a");
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, begin, end);
    fprintf(csv_output, "KernelMatrixAdd, %d, %d, %f", LENGTH, BLOCK_SIZE * BLOCK_SIZE, elapsed_time);
    fclose(csv_output);

    hipMemcpy2D(result, WIDTH * sizeof(float), dev_result, pitch, WIDTH, HEIGHT, hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_result);

    free(A);
    free(B);
    free(result);
}

